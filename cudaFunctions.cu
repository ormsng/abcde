#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <math.h>
#include "Proto.h"
#include "Structs.h"

__global__ void calculatePoint(Axis *axisArr, Point *pointArr, int numElements, double t)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        pointArr[i].x = ((axisArr[i].x2 - axisArr[i].x1) / 2) * sin(t * PI / 2) + (axisArr[i].x2 + axisArr[i].x1) / 2;
        pointArr[i].y = axisArr[i].a * pointArr[i].x + axisArr[i].b;
    }
}

__global__ void ProximityCriteria(int rank, int chunkSize, int *flags, Point *pointArr, int numElements, float D, int K)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x; 
    int index = tid + rank * chunkSize; 
    int counter = 0;
    if (index < numElements)
    {
        Point p1 = pointArr[index]; 
        for (int i = 0; i < numElements && counter < K; i++) 
        {
            if (index == i)
                continue;
            Point p2 = pointArr[i]; 
            if (sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2)) <= D)
                counter++;
        }
        if (counter == K)
            flags[tid] = 1;
    }
}

int computePointsOnGPU(Axis *axisArr, Point *pointArr, int numElements, double t)
{
    hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(Axis);
    Axis *d_Axis;
    err = hipMalloc((void **)&d_Axis, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_Axis, axisArr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    size = numElements * sizeof(Point);
    Point *d_Points;
    err = hipMalloc((void **)&d_Points, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 100;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    calculatePoint<<<blocksPerGrid, threadsPerBlock>>>(d_Axis, d_Points, numElements, t); 
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(pointArr, d_Points, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_Axis);
    hipFree(d_Points);
    return 1;
}

void checkProximityCriteriaOnGPU(int rank, Point *allPoints, int N,int* flags, int chunkSize, float D, int K)
{
    hipError_t err = hipSuccess;

    size_t size = N * sizeof(Point);
    Point *d_Points;
    err = hipMalloc((void **)&d_Points, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_Points, allPoints, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        hipFree(d_Points);
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int *d_Flags;
    size = chunkSize * sizeof(int);
    err = hipMalloc((void **)&d_Flags, size);
    if (err != hipSuccess)
    {   hipFree(d_Flags);
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset(d_Flags, 0, size);
    if (err != hipSuccess)
    {   
        hipFree(d_Flags);
        hipFree(d_Points);
        fprintf(stderr, "Failed to set device memory to zero- %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int threadsPerBlock = 100;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    ProximityCriteria<<<blocksPerGrid, threadsPerBlock>>>(rank, chunkSize, d_Flags, d_Points, N, D, K);//set flagArr

    err = hipMemcpy(flags, d_Flags, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        hipFree(d_Flags);
        hipFree(d_Points);
        free(flags);
        fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipFree(d_Flags);
    hipFree(d_Points);
}
